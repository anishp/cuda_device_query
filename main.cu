#include <stdio.h>
#include <conio.h>
#include <hip/hip_runtime.h>

int main(int argc, char** argv[])
{
	int num_devices = 0;
	hipDeviceProp_t mydevice;

	hipError_t err = hipGetDeviceCount(&num_devices);
	if(err != hipSuccess)	printf("%sn", hipGetErrorString(err));

	printf("Cuda devices: %d\n\n", num_devices);

	for(int i=0; i<num_devices; i++)
	{
		err = hipGetDeviceProperties(&mydevice, 0);
		if(err != hipSuccess) printf("%sn", hipGetErrorString(err));
		
		printf("\tCuda device %d: %s\n", i, mydevice.name);
	}

	getchar();
	return 0;
}