#include <stdio.h>
#include <conio.h>
#include <hip/hip_runtime.h>

int main(int argc, char** argv[])
{
	int num_devices = 0;
	hipDeviceProp_t mydevice;

	hipError_t err = hipGetDeviceCount(&num_devices);
	if(err != hipSuccess)	printf("%sn", hipGetErrorString(err));

	printf("Cuda devices: %d\n\n", num_devices);

	for(int i=0; i<num_devices; i++)
	{
		err = hipGetDeviceProperties(&mydevice, 0);
		if(err != hipSuccess) printf("%sn", hipGetErrorString(err));
		
		printf("  Cuda device %d: %s\n", i, mydevice.name);
		printf("  Multiprocessors: %d\n", mydevice.multiProcessorCount);
		printf("  Device Clock: %d KHz\n", mydevice.clockRate);
		printf("  Memory Clock Rate (KHz): %d\n", mydevice.memoryClockRate);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*mydevice.memoryClockRate*(mydevice.memoryBusWidth/8)/1.0e6);
	}

	getchar();
	return 0;
}